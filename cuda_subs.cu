#include "hip/hip_runtime.h"
#include "cuda_subs.h"

hipDoubleComplex  *dev_rhophon;
hipDoubleComplex  *dev_rhotot;
hipDoubleComplex  *dev_rhonew;
hipDoubleComplex  *dev_rhoaux;
hipDoubleComplex  *dev_Drho;
hipDoubleComplex  *dev_Htot1;
hipDoubleComplex  *dev_Htot2;
hipDoubleComplex  *dev_Htot3;
hipDoubleComplex  *dev_mutot;
hipDoubleComplex  *dev_dvdx;
double           *dev_vbath;
double           *dev_fb;
double           *dev_xi;
double           *dev_vi;
double           *dev_ki;
double           *dev_xf;
double           *dev_vf;
double           *dev_xh;
UNINT             Ncores1;
UNINT             Ncores2;

//##############################################################################
// This function build the Hamiltonian without CEED:
// H_in  = H_e + H_phon + H_e-phon - This matrix is the same always.
// H_out = H_in + E * mu + V_bath
__global__ void update_H_tot(hipDoubleComplex *H_out, hipDoubleComplex *H_in,
                             hipDoubleComplex *mu_tot,
                             double *v_bath_mat, double *fb_vec,
                             double sum_xi, double Efield, int n_el,
                             int n_phon, int np_levels, int n_tot){

   hipDoubleComplex aux1;
   hipDoubleComplex aux2;
   hipDoubleComplex aux3;

   int ind  = threadIdx.x + blockIdx.x * blockDim.x;
   int dim2 = n_tot * n_tot;

   if (ind < dim2){
      int i1   = ind / n_tot;
      int i_e  = i1 / (n_phon*np_levels);

      H_out[ind] = make_hipDoubleComplex(0.0e0,0.0e0);

      if ( ind == i1 + i1*n_tot ){
         aux1       = make_hipDoubleComplex(fb_vec[i_e] * sum_xi, 0.0e0);
         H_out[ind] = hipCadd(H_in[ind],aux1);
      }

      aux1 = make_hipDoubleComplex(Efield, 0.0e0);
      aux1 = hipCmul(aux1, mu_tot[ind]);

      aux2 = make_hipDoubleComplex(sum_xi*v_bath_mat[ind], 0.0e0);

      aux3 = hipCadd(aux1,aux2);
      H_out[ind] = hipCadd(H_out[ind], aux3);
   }
   return;
}
//##############################################################################
//This function extract the diagonal terms of the matrix matA in vecA
__global__ void get_diag(hipDoubleComplex *matA, hipDoubleComplex *vecA,
                         int n_tot){

   int ind  = threadIdx.x + blockIdx.x * blockDim.x;
   int dim2 = n_tot  * n_tot;
   int i1   = ind / n_tot;
   if ((ind == i1 + i1*n_tot) && (ind < dim2)){
      vecA[i1] = matA[ind];
   }
   return;
}
//##############################################################################
__global__ void build_rhophon(hipDoubleComplex *rho_tot,
                              hipDoubleComplex *rho_phon, int n_el ,int n_phon,
                              int np_levels, int n_tot){

   int ind1 = threadIdx.x + blockIdx.x * blockDim.x;
   int dim1 = n_phon * np_levels;
   int dim2 = dim1 * dim1;

   if (ind1 < dim2){
      int jj = ind1/dim1;
      int ii = ind1 - jj * dim1;
      rho_phon[ind1] = make_hipDoubleComplex(0.0e0, 0.0e0);
      for (int kk=0; kk<n_el; kk++){
         int ind2 = (ii + kk * dim1) + (jj + kk * dim1) * n_tot;
         rho_phon[ind1] = hipCadd(rho_tot[ind2], rho_phon[ind1]);
      }
   }
   return;
}
//##############################################################################
__global__ void move_x(double *xi_vec, double *vi_vec, double *xf_vec,
                           double dt, int n_bath){
   int ind        = threadIdx.x + blockIdx.x * blockDim.x;
   if (ind < n_bath){
      xf_vec[ind]  = xi_vec[ind] + vi_vec[ind] * dt;
   }
   return;
}
//##############################################################################
__global__ void get_partial_sum(double *xi_vec, double *sum_vec, int n_bath){
   __shared__ double cache[Nthreads];
   int ind        = threadIdx.x + blockIdx.x * blockDim.x;
   int cacheIndex = threadIdx.x;

   cache[cacheIndex] = 0.0e0;
   if (ind < n_bath){
      cache[cacheIndex] = xi_vec[ind];
      __syncthreads();
      int ii = blockDim.x/2;
      while (ii != 0) {
         if (cacheIndex < ii){
            cache[cacheIndex] += cache[cacheIndex + ii];
         }
         __syncthreads();
         ii /= 2;
      }
      if (cacheIndex == 0){
         sum_vec[blockIdx.x] = cache[0];
      }
   }
   return;
}
//##############################################################################
__global__ void get_partial_Ek(double *vi_vec, double *sum_vec, int n_bath){
   __shared__ double cache[Nthreads];
   int ind        = threadIdx.x + blockIdx.x * blockDim.x;
   int cacheIndex = threadIdx.x;

   cache[cacheIndex] = 0.0e0;
   if (ind < n_bath){
      cache[cacheIndex] = vi_vec[ind]*vi_vec[ind];
      __syncthreads();
      int ii = blockDim.x/2;
      while (ii != 0) {
         if (cacheIndex < ii){
            cache[cacheIndex] += cache[cacheIndex + ii];
         }
         __syncthreads();
         ii /= 2;
      }
      if (cacheIndex == 0){
         sum_vec[blockIdx.x] = cache[0];
      }
   }
   return;
}
//##############################################################################
__global__ void move_v(double *xi_vec, double *vi_vec, double *ki_vec,
                       double *vf_vec, double qforce, double dt,
                       int n_bath){

   int ind  = threadIdx.x + blockIdx.x * blockDim.x;
   if (ind < n_bath){
      double ai = - ki_vec[ind] * xi_vec[ind] + qforce;
      vf_vec[ind]  = vi_vec[ind] + ai * dt;
   }
}
//##############################################################################
__global__ void update_mat(hipDoubleComplex *matA, hipDoubleComplex *matB,
                           int dim){
   int ind  = threadIdx.x + blockIdx.x * blockDim.x;
   int dim2 = dim * dim;
   if (ind < dim2){
      matA[ind] = matB[ind];
   }
   return;
}
//##############################################################################
__global__ void update_vec(double *vecA, double *vecB, int dim){
   int ind  = threadIdx.x + blockIdx.x * blockDim.x;
   if (ind < dim){
      vecA[ind] = vecB[ind];
   }
   return;
}
//##############################################################################
void init_cuda(complex<double> *H_tot, complex<double> *mu_tot,
               double *v_bath_mat, double *fb_vec, double *xi_vec,
               double *vi_vec, double *ki_vec,
               complex<double> *rho_tot,
               complex<double> *rho_phon, complex<double> *dVdX_mat,
               UNINT n_el, UNINT n_phon, UNINT np_levels, UNINT n_tot,
               UNINT n_bath){

   double gaux = (double) (n_tot*n_tot);
   double taux = (double) Nthreads;

   Ncores1 = (UNINT) ceil(gaux/taux);
   gaux    = (double) (n_bath);
   Ncores2 = (UNINT) ceil(gaux/taux);

   int dimaux  = n_phon * n_phon * np_levels * np_levels;

   hipMalloc((void**) &dev_Htot1  , n_tot*n_tot*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_Htot2  , n_tot*n_tot*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_Htot3  , n_tot*n_tot*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_mutot  , n_tot*n_tot*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_rhotot , n_tot*n_tot*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_rhonew , n_tot*n_tot*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_rhoaux , n_tot*n_tot*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_Drho   , n_tot*n_tot*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_rhophon, dimaux*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_dvdx   , dimaux*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_vbath  , n_tot*n_tot*sizeof(double));
   hipMalloc((void**) &dev_fb     , n_el*sizeof(double));
   hipMalloc((void**) &dev_xi     , n_bath*sizeof(double));
   hipMalloc((void**) &dev_vi     , n_bath*sizeof(double));
   hipMalloc((void**) &dev_ki     , n_bath*sizeof(double));
   hipMalloc((void**) &dev_xf     , n_bath*sizeof(double));
   hipMalloc((void**) &dev_vf     , n_bath*sizeof(double));
   hipMalloc((void**) &dev_xh     , n_bath*sizeof(double));

   hipMemcpy(dev_Htot1, H_tot, n_tot*n_tot*sizeof(hipDoubleComplex),
              hipMemcpyHostToDevice);
   hipMemcpy(dev_mutot, mu_tot, n_tot*n_tot*sizeof(hipDoubleComplex),
              hipMemcpyHostToDevice);
   hipMemcpy(dev_rhotot, rho_tot, n_tot*n_tot*sizeof(hipDoubleComplex),
              hipMemcpyHostToDevice);
   hipMemcpy(dev_rhophon, rho_phon, dimaux*sizeof(hipDoubleComplex),
              hipMemcpyHostToDevice);
   hipMemcpy(dev_dvdx, dVdX_mat, dimaux*sizeof(hipDoubleComplex),
              hipMemcpyHostToDevice);
   hipMemcpy(dev_vbath, v_bath_mat, n_tot*n_tot*sizeof(double),
              hipMemcpyHostToDevice);
   hipMemcpy(dev_fb, fb_vec, n_el*sizeof(double), hipMemcpyHostToDevice);
   hipMemcpy(dev_xi, xi_vec, n_bath*sizeof(double), hipMemcpyHostToDevice);
   hipMemcpy(dev_vi, vi_vec, n_bath*sizeof(double), hipMemcpyHostToDevice);
   hipMemcpy(dev_ki, ki_vec, n_bath*sizeof(double), hipMemcpyHostToDevice);

   return;
}
//##############################################################################
void free_cuda_memory(){

   hipFree(dev_Htot1);
   hipFree(dev_Htot2);
   hipFree(dev_Htot3);
   hipFree(dev_mutot);
   hipFree(dev_rhotot);
   hipFree(dev_rhonew);
   hipFree(dev_rhoaux);
   hipFree(dev_Drho);
   hipFree(dev_rhophon);
   hipFree(dev_dvdx);
   hipFree(dev_vbath);
   hipFree(dev_fb);
   hipFree(dev_ki);
   hipFree(dev_xi);
   hipFree(dev_xf);
   hipFree(dev_xh);
   hipFree(dev_vi);
   hipFree(dev_vf);

   return;
}
//##############################################################################
void matmul_cublas(hipDoubleComplex *dev_A, hipDoubleComplex *dev_B,
                   hipDoubleComplex *dev_C, int dim){

   const hipDoubleComplex alf = make_hipDoubleComplex(1.0,0.0);
   const hipDoubleComplex bet = make_hipDoubleComplex(0.0, 0.0);
   const hipDoubleComplex *alpha = &alf;
   const hipDoubleComplex *beta = &bet;
// Create a handle for CUBLAS
   hipblasHandle_t handle;
   hipblasCreate(&handle);
// Do the actual multiplication
    hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, alpha, dev_A,
                dim, dev_B, dim, beta, dev_C, dim);
// Destroy the handle
    hipblasDestroy(handle);
    return;

}
//##############################################################################
void commute_cuda(hipDoubleComplex *dev_A, hipDoubleComplex *dev_B,
                  hipDoubleComplex *dev_C, int dim, const hipDoubleComplex alf){

   const hipDoubleComplex bet1 = make_hipDoubleComplex(0.0, 0.0);
   const hipDoubleComplex bet2 = make_hipDoubleComplex(-1.0, 0.0);
   const hipDoubleComplex *alpha = &alf;
   const hipDoubleComplex *beta1 = &bet1;
   const hipDoubleComplex *beta2 = &bet2;
// Create a handle for CUBLAS
   hipblasHandle_t handle;
   hipblasCreate(&handle);
// Computing B.A
   hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, alpha, dev_B,
               dim, dev_A, dim, beta1, dev_C, dim);
// Computing A.B - B.A
   hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, alpha, dev_A,
               dim, dev_B, dim, beta2, dev_C, dim);
// Destroy the handle
   hipblasDestroy(handle);
   return;
}
//##############################################################################
void matadd_cublas(hipDoubleComplex *dev_A, hipDoubleComplex *dev_B,
                   hipDoubleComplex *dev_C, int dim, const hipDoubleComplex alf,
                   const hipDoubleComplex bet){

   const hipDoubleComplex *alpha = &alf;
   const hipDoubleComplex *beta = &bet;
// Create a handle for CUBLAS
   hipblasHandle_t handle;
   hipblasCreate(&handle);
// Do the actual multiplication
    hipblasZgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, alpha, dev_A,
                dim, beta ,dev_B, dim, dev_C, dim);
// Destroy the handle
    hipblasDestroy(handle);
    return;
}
//##############################################################################
double get_trace_cuda(hipDoubleComplex *dev_A, UNINT dim){

   // hipDoubleComplex aux1= make_hipDoubleComplex(0.0e0, 0.0e0);
   complex<double> aux1;
   double          aux2;
   complex<double> aux_vec[dim];
   hipDoubleComplex *dev_vec;

   hipMalloc((void**) &dev_vec, dim * sizeof(hipDoubleComplex));

   get_diag<<<Ncores1, Nthreads>>>(dev_A, dev_vec, dim);

   hipMemcpy(aux_vec, dev_vec, dim*sizeof(hipDoubleComplex),
              hipMemcpyDeviceToHost);

   for(int ii=0;ii<dim;ii++){
      aux1 += aux_vec[ii];
   }

   aux2 = aux1.real();

   hipFree(dev_vec);
   return aux2;
}
//##############################################################################
void include_Hceed_cuda(hipDoubleComplex *dev_Hout, hipDoubleComplex *dev_Hin,
                        hipDoubleComplex *dev_mu, hipDoubleComplex *dev_rhoin,
                        double a_ceed, int n_tot){

   int dim2 = n_tot*n_tot;
   double dmu2;
   hipDoubleComplex *dev_aux1, *dev_aux2, *dev_Hceed;
   const hipDoubleComplex alf  = make_hipDoubleComplex(1.0,0.0);

   hipMalloc((void**) &dev_aux1, dim2 * sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_aux2, dim2 * sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_Hceed, dim2 * sizeof(hipDoubleComplex));

   commute_cuda(dev_mu, dev_Hin, dev_Hceed, n_tot, alf);
   commute_cuda(dev_Hceed, dev_Hin, dev_aux1, n_tot, alf);
   matmul_cublas(dev_rhoin, dev_aux1, dev_aux2, n_tot);

   dmu2 = get_trace_cuda(dev_aux2, n_tot);

   const hipDoubleComplex bet = make_hipDoubleComplex(0.0, a_ceed*dmu2);

   matadd_cublas(dev_Hin, dev_Hceed, dev_Hout, n_tot, alf, bet);

   hipFree(dev_aux1);
   hipFree(dev_aux2);
   hipFree(dev_Hceed);

   return;
}
//##############################################################################
double get_Qforces_cuda(hipDoubleComplex *dev_rhoin ,double *fb_vec,
                        UNINT n_el, UNINT n_phon, UNINT np_levels, UNINT n_tot){


   UNINT            dim1   = np_levels * n_phon;
   UNINT            dim2   = dim1 * dim1;
   double           qforce = 0.0e0;
   complex<double>  aux_vec[n_tot];
   hipDoubleComplex *dev_vec, *dev_mat;

   hipMalloc((void**) &dev_vec, n_tot * sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_mat, dim2 * sizeof(hipDoubleComplex));

   get_diag<<<Ncores1, Nthreads>>>(dev_rhoin, dev_vec, n_tot);

   hipMemcpy(aux_vec, dev_vec, n_tot*sizeof(hipDoubleComplex),
              hipMemcpyDeviceToHost);

   for (int kk=0; kk<n_el; kk++){
   for (int ii=0; ii<dim1; ii++){
      qforce += -aux_vec[ii+kk*dim1].real() * fb_vec[kk];
   }
   }

   build_rhophon<<<Ncores1, Nthreads>>>(dev_rhoin, dev_rhophon, n_el , n_phon,
                                        np_levels, n_tot);
   matmul_cublas(dev_rhophon, dev_dvdx, dev_mat, dim1);

   get_diag<<<Ncores1, Nthreads>>>(dev_mat, dev_vec, dim1);

   hipMemcpy(aux_vec, dev_vec, n_tot*sizeof(hipDoubleComplex),
              hipMemcpyDeviceToHost);

   for (int ii=0; ii<dim1; ii++){
      qforce += -aux_vec[ii].real();
   }

   hipFree(dev_vec);
   hipFree(dev_mat);
   return qforce;
}
//##############################################################################
void runge_kutta_propagator_cuda(double mass_bath, double a_ceed, double dt,
                                 double Efield, double *fb_vec, int tt,
                                 UNINT n_el, UNINT n_phon, UNINT np_levels,
                                 UNINT n_tot, UNINT n_bath){

   const hipDoubleComplex alf1 = make_hipDoubleComplex(0.0e0, -0.5*dt);
   const hipDoubleComplex alf2 = make_hipDoubleComplex(0.0e0, -dt);
   const hipDoubleComplex alf3 = make_hipDoubleComplex(1.0e0, 0.0e0);
   double *dev_partialvec;
   double  partialvec[Ncores2];
   double  sum_xi;
   double  dth    = 0.5e0 * dt;
   double Efield_t;
   double qforce;
   double time = dt * tt;

   hipMalloc((void**) &dev_partialvec, Ncores2*sizeof(double));

   //Calculating the sum of all the coordintes of the bath----------------------
   get_partial_sum<<<Ncores2, Nthreads>>>(dev_xi, dev_partialvec, n_bath);

   hipMemcpy(partialvec, dev_partialvec, Ncores2*sizeof(double),
              hipMemcpyDeviceToHost);

   sum_xi = 0.0e0;
   for (int ii=1; ii<Ncores2; ii++){
      sum_xi += partialvec[ii];
   }
   //---------------------------------------------------------------------------

   Efield_t = Efield * exp(-pow(((time-10.0)/0.2),2.0));

   //Building the new Hamiltonian at time = t ----------------------------------
   update_H_tot<<<Ncores1, Nthreads>>>(dev_Htot2, dev_Htot1, dev_mutot,
                                       dev_vbath, dev_fb, sum_xi, Efield_t,
                                       n_el, n_phon, np_levels, n_tot);
   //Including CEED Hamiltonian:
   include_Hceed_cuda(dev_Htot3, dev_Htot2, dev_mutot, dev_rhotot, a_ceed,
                      n_tot);
   //---------------------------------------------------------------------------

   //Calculating rho(t+dt/2) using LvN------------------------------------------
   commute_cuda(dev_Htot3, dev_rhotot, dev_Drho, n_tot, alf3);
   matadd_cublas(dev_rhotot, dev_Drho, dev_rhoaux, n_tot, alf3, alf1);
   //---------------------------------------------------------------------------
   //Calculating x(t+dt/2) and v(t+dt/2) using the Quantum forces --------------
   qforce = get_Qforces_cuda(dev_rhotot , fb_vec, n_el, n_phon, np_levels,
                             n_tot);
   qforce = qforce/mass_bath;
   move_x<<<Ncores2, Nthreads>>>(dev_xi, dev_vi, dev_xh, dth, n_bath);


   move_v<<<Ncores2, Nthreads>>>(dev_xi, dev_vi, dev_ki, dev_vf, qforce, dth,
                                 n_bath);
   //---------------------------------------------------------------------------
   //Hencefort we repeat everything to obtain everything in t + dt -------------

   get_partial_sum<<<Ncores2, Nthreads>>>(dev_xh, dev_partialvec, n_bath);

   hipMemcpy(partialvec, dev_partialvec, Ncores2*sizeof(double),
   hipMemcpyDeviceToHost);

   sum_xi = 0.0e0;
   for (int ii=1; ii<Ncores2; ii++){
      sum_xi += partialvec[ii];
   }

   Efield_t = Efield * exp(-pow(((time+dth-10.0)/0.2),2.0));

   update_H_tot<<<Ncores1, Nthreads>>>(dev_Htot2, dev_Htot1, dev_mutot,
                                       dev_vbath, dev_fb, sum_xi, Efield_t,
                                       n_el, n_phon, np_levels, n_tot);

   include_Hceed_cuda(dev_Htot3, dev_Htot2, dev_mutot, dev_rhoaux, a_ceed,
                      n_tot);

   commute_cuda(dev_Htot3, dev_rhoaux, dev_Drho, n_tot, alf3);
   matadd_cublas(dev_rhotot, dev_Drho, dev_rhonew, n_tot, alf3, alf2);

   qforce = get_Qforces_cuda(dev_rhoaux , fb_vec, n_el, n_phon, np_levels,
                             n_tot);
   qforce = qforce/mass_bath;
   move_x<<<Ncores2, Nthreads>>>(dev_xi, dev_vf, dev_xf, dt, n_bath);

   move_v<<<Ncores2, Nthreads>>>(dev_xh, dev_vi, dev_ki, dev_vf, qforce, dt,
                                 n_bath);
   //---------------------------------------------------------------------------
   //We update rho, x and v:
   update_mat<<<Ncores1, Nthreads>>>(dev_rhotot, dev_rhonew, n_tot);
   update_vec<<<Ncores2, Nthreads>>>(dev_xi, dev_xf, n_bath);
   update_vec<<<Ncores2, Nthreads>>>(dev_vi, dev_vf, n_bath);

   hipFree(dev_partialvec);

   return;
}
//##############################################################################
void calcrhophon(hipDoubleComplex *dev_rhoin, int n_el, int n_phon,
                 int np_levels, int n_tot){
   build_rhophon<<<Ncores1, Nthreads>>>(dev_rhoin, dev_rhophon, n_el , n_phon,
                                        np_levels, n_tot);
   return;
}
//##############################################################################
void getingmat(complex<double> *matA, hipDoubleComplex *dev_A, int n_tot){

   hipMemcpy(matA, dev_A, n_tot*n_tot*sizeof(hipDoubleComplex),
              hipMemcpyDeviceToHost);
   return;
}
//##############################################################################
void getting_printing_info(double *Ener, double *mu, complex<double> *tr_rho,
                           double *Ek_bath, UNINT n_tot, UNINT n_bath){

   int dim2 = n_tot * n_tot;
   hipDoubleComplex *dev_aux1;
   hipDoubleComplex *dev_vec;
   double *dev_partialvec;
   double  partialvec[Ncores2];

   hipMalloc((void**) &dev_aux1, dim2 * sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_vec, n_tot * sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_partialvec, Ncores2*sizeof(double));

   matmul_cublas(dev_rhotot, dev_Htot1, dev_aux1, n_tot);
   *Ener = get_trace_cuda(dev_aux1, n_tot);

   matmul_cublas(dev_rhotot, dev_mutot, dev_aux1, n_tot);
   *mu = get_trace_cuda(dev_aux1, n_tot);

   get_diag<<<Ncores1, Nthreads>>>(dev_rhotot, dev_vec, n_tot);

   hipMemcpy(tr_rho, dev_vec, n_tot*sizeof(hipDoubleComplex),
   hipMemcpyDeviceToHost);

   get_partial_Ek<<<Ncores2, Nthreads>>>(dev_vi, dev_partialvec, n_bath);
   hipMemcpy(partialvec, dev_partialvec, Ncores2*sizeof(double),
              hipMemcpyDeviceToHost);

   *Ek_bath = 0.0e0;
   for (int ii=1; ii<Ncores2; ii++){
      *Ek_bath += 0.5e0 * partialvec[ii];
   }

   hipFree(dev_vec);
   hipFree(dev_aux1);
   hipFree(dev_partialvec);

   return;
}
//##############################################################################
