#include "hip/hip_runtime.h"
#include "cuda_subs.h"

hipDoubleComplex  *dev_rhoelec;
hipDoubleComplex  *dev_rhophon;
hipDoubleComplex  *dev_rhotot;
hipDoubleComplex  *dev_rhonew;
hipDoubleComplex  *dev_Htot1;
hipDoubleComplex  *dev_Htot2;
hipDoubleComplex  *dev_Htot3;
hipDoubleComplex  *dev_mutot;
double           *dev_vbath;
double           *dev_fb;
UNINT             Ncores;

//##############################################################################
__global__ void update_H_tot(hipDoubleComplex *H_out, hipDoubleComplex *H_tot,
                             hipDoubleComplex *mu_tot,
                             double *v_bath_mat, double *fb_vec,
                             double sum_xi, double Efield, int n_el,
                             int n_phon, int np_levels, int n_tot){

   hipDoubleComplex aux1;
   hipDoubleComplex aux2;
   hipDoubleComplex aux3;

   int ind  = threadIdx.x + blockIdx.x * blockDim.x;
   int dim2 = n_tot * n_tot;
   int i1   = ind / n_tot;
   int i_e  = i1 / (n_phon*np_levels);

   if (ind < dim2){
      if ( ind == i1 + i1*n_tot ){
         aux1       = make_hipDoubleComplex(fb_vec[i_e] * sum_xi, 0.0e0);
         H_out[ind] = hipCadd(H_tot[ind],aux1);
      }

      aux1 = make_hipDoubleComplex(Efield, 0.0e0);
      aux1 = hipCmul(aux1, mu_tot[ind]);

      aux2 = make_hipDoubleComplex(sum_xi*v_bath_mat[ind], 0.0e0);

      aux3 = hipCadd(aux1,aux2);
      H_out[ind] = hipCadd(H_out[ind], aux3);
   }

   return;
}
//##############################################################################
__global__ void get_diag(hipDoubleComplex *matA, hipDoubleComplex *vecA,
                         int n_tot){

   int ind  = threadIdx.x + blockIdx.x * blockDim.x;
   int dim2 = n_tot  * n_tot;
   int i1   = ind / n_tot;

   if ((ind == i1 + i1*n_tot) && (ind < dim2)){
      vecA[i1] = matA[ind];
   }

   return;
}
//##############################################################################
__global__ void build_Hceed(hipDoubleComplex *H_out, hipDoubleComplex *H_in,
                            hipDoubleComplex *H_ceed, double dmu2,
                            double a_ceed, int n_tot){

   int ind  = threadIdx.x + blockIdx.x * blockDim.x;
   int dim2 = n_tot * n_tot;

   if (ind < dim2){
      hipDoubleComplex aux1 = make_hipDoubleComplex(0.0e0, a_ceed*dmu2);
      hipDoubleComplex aux2;
      aux2       = hipCmul(aux1,H_ceed[ind]);
      H_out[ind] = hipCadd(H_in[ind], aux2);
   }

   return;
}
//##############################################################################
void init_cuda(complex<double> *H_tot, complex<double> *mu_tot,
               double *v_bath_mat, double *fb_vec, complex<double> *rho_tot,
               UNINT n_el, UNINT n_phon, UNINT n_tot){

   double gaux = (double) (n_tot*n_tot);
   double taux = (double) Nthreads;

   Ncores = (UNINT) ceil(gaux/taux);

   hipMalloc((void**) &dev_Htot1  , n_tot*n_tot*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_Htot2  , n_tot*n_tot*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_Htot3  , n_tot*n_tot*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_mutot  , n_tot*n_tot*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_rhotot , n_tot*n_tot*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_rhonew , n_tot*n_tot*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_vbath  , n_tot*n_tot*sizeof(double));
   hipMalloc((void**) &dev_fb     , n_el*sizeof(double));

   hipMemcpy(dev_Htot1, H_tot, n_tot*n_tot*sizeof(hipDoubleComplex),
              hipMemcpyHostToDevice);
   hipMemcpy(dev_Htot2, H_tot, n_tot*n_tot*sizeof(hipDoubleComplex),
              hipMemcpyHostToDevice);
   hipMemcpy(dev_Htot3, H_tot, n_tot*n_tot*sizeof(hipDoubleComplex),
              hipMemcpyHostToDevice);
   hipMemcpy(dev_mutot, mu_tot, n_tot*n_tot*sizeof(hipDoubleComplex),
              hipMemcpyHostToDevice);
   hipMemcpy(dev_rhotot, rho_tot, n_tot*n_tot*sizeof(hipDoubleComplex),
              hipMemcpyHostToDevice);
   hipMemcpy(dev_rhonew, rho_tot, n_tot*n_tot*sizeof(hipDoubleComplex),
              hipMemcpyHostToDevice);
   hipMemcpy(dev_vbath, v_bath_mat, n_tot*n_tot*sizeof(double),
              hipMemcpyHostToDevice);
   hipMemcpy(dev_fb, fb_vec, n_el*sizeof(double), hipMemcpyHostToDevice);

   return;
}
//##############################################################################
void free_cuda_memory(){

   hipFree(dev_Htot1);
   hipFree(dev_Htot2);
   hipFree(dev_Htot3);
   hipFree(dev_mutot);
   hipFree(dev_rhotot);
   hipFree(dev_rhonew);
   hipFree(dev_vbath);
   hipFree(dev_fb);

   return;
}
//##############################################################################
void matmul_cublas(hipDoubleComplex *dev_A, hipDoubleComplex *dev_B,
                   hipDoubleComplex *dev_C, int dim){

   const hipDoubleComplex alf = make_hipDoubleComplex(1.0,0.0);
   const hipDoubleComplex bet = make_hipDoubleComplex(0.0, 0.0);
   const hipDoubleComplex *alpha = &alf;
   const hipDoubleComplex *beta = &bet;

// Create a handle for CUBLAS
   hipblasHandle_t handle;
   hipblasCreate(&handle);

// Do the actual multiplication

    hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, alpha, dev_A,
                dim, dev_B, dim, beta, dev_C, dim);

// Destroy the handle
    hipblasDestroy(handle);

    return;

}
//##############################################################################
void commute_cuda(hipDoubleComplex *dev_A, hipDoubleComplex *dev_B,
                  hipDoubleComplex *dev_C, int dim){

   const hipDoubleComplex alf  = make_hipDoubleComplex(1.0,0.0);
   const hipDoubleComplex bet1 = make_hipDoubleComplex(0.0, 0.0);
   const hipDoubleComplex bet2 = make_hipDoubleComplex(-1.0, 0.0);
   const hipDoubleComplex *alpha = &alf;
   const hipDoubleComplex *beta1 = &bet1;
   const hipDoubleComplex *beta2 = &bet2;

// Create a handle for CUBLAS
   hipblasHandle_t handle;
   hipblasCreate(&handle);

// Computing B.A

   hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, alpha, dev_B,
               dim, dev_A, dim, beta1, dev_C, dim);

// Computing A.B - B.A
   hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, alpha, dev_A,
               dim, dev_B, dim, beta2, dev_C, dim);

// Destroy the handle
   hipblasDestroy(handle);

   return;
}
//##############################################################################
double get_trace_cuda(hipDoubleComplex *dev_A, UNINT dim){

   // hipDoubleComplex aux1= make_hipDoubleComplex(0.0e0, 0.0e0);
   complex<double> aux1;
   double          aux2;
   complex<double> aux_vec[dim];
   hipDoubleComplex *dev_vec;

   hipMalloc((void**) &dev_vec, dim * sizeof(hipDoubleComplex));

   get_diag<<<Ncores, Nthreads>>>(dev_A, dev_vec, dim);

   hipMemcpy(aux_vec, dev_vec, dim*sizeof(hipDoubleComplex),
              hipMemcpyDeviceToHost);

   for(int ii=0;ii<dim;ii++){
      aux1 += aux_vec[ii];
   }

   aux2 = aux1.real();

   hipFree(dev_vec);
   return aux2;
}
//##############################################################################
void include_Hceed_cuda(hipDoubleComplex *dev_Hout, hipDoubleComplex *dev_Hin,
                        hipDoubleComplex *dev_Hceed,
                        hipDoubleComplex *dev_mu, hipDoubleComplex *dev_rhoaux,
                        double a_ceed, int n_tot){

   int dim2 = n_tot*n_tot;
   double dmu2;
   hipDoubleComplex *dev_aux1, *dev_aux2;

   hipMalloc((void**) &dev_aux1, dim2 * sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_aux2, dim2 * sizeof(hipDoubleComplex));

   commute_cuda(dev_Hin, dev_mu, dev_Hceed, n_tot);
   commute_cuda(dev_Hceed, dev_Hin, dev_aux1, n_tot);
   matmul_cublas(dev_rhoaux, dev_aux1, dev_aux2, n_tot);

   dmu2 = get_trace_cuda(dev_aux2, n_tot);

   build_Hceed<<<Ncores, Nthreads>>>(dev_Hout, dev_Hin, dev_Hceed, dmu2,
                                     a_ceed, n_tot);

   hipFree(dev_aux1);
   hipFree(dev_aux2);

   return;
}
//##############################################################################
void getingmat(complex<double> *matA, hipDoubleComplex *dev_A, int n_tot){

   hipMemcpy(matA, dev_A, n_tot*n_tot*sizeof(hipDoubleComplex),
              hipMemcpyDeviceToHost);
   return;
}
//
