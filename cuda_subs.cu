#include "hip/hip_runtime.h"
#include "cuda_subs.h"

hipDoubleComplex  *dev_rhophon;
hipDoubleComplex  *dev_rhotot;
hipDoubleComplex  *dev_rhonew;
hipDoubleComplex  *dev_rhoaux;
hipDoubleComplex  *dev_Drho;
hipDoubleComplex  *dev_Htot1;
hipDoubleComplex  *dev_Htot2;
hipDoubleComplex  *dev_Htot3;
hipDoubleComplex  *dev_mutot;
hipDoubleComplex  *dev_dvdx;
double           *dev_vbath;
double           *dev_fb;
double           *dev_xi;
double           *dev_vi;
double           *dev_ki;
double           *dev_xf;
double           *dev_vf;
double           *dev_xh;
double           *dev_etaL_ke;
double           *dev_lambdaL_ke;
double           *dev_etaS_ke;
double           *dev_lambdaS_ke;
double           *dev_ke_del1;
double           *dev_ke_del2;
double           *dev_Nphon_ke;
int              *dev_keind_j;
int              *dev_keind_k;
UNINT             Ncores1;
UNINT             Ncores2;
UNINT             Ncores3;

//##############################################################################
// This function build the Hamiltonian without CEED:
// H_in  = H_e + H_phon + H_e-phon - This matrix is the same always.
// H_out = H_in + E * mu + V_bath
__global__ void update_H_tot(hipDoubleComplex *H_out, hipDoubleComplex *H_in,
                             hipDoubleComplex *mu_tot,
                             double *v_bath_mat, double *fb_vec,
                             double sum_xi, double Efield, int n_el,
                             int n_phon, int np_levels, int n_tot){

   hipDoubleComplex aux1;
   hipDoubleComplex aux2;
   hipDoubleComplex aux3;

   int ind  = threadIdx.x + blockIdx.x * blockDim.x;
   int dim2 = n_tot * n_tot;

   if (ind < dim2){
      int i1   = ind / n_tot;
      int i_e  = i1 / (n_phon*np_levels);

      H_out[ind] = H_in[ind];

      if ( ind == i1 + i1*n_tot ){
         aux1       = make_hipDoubleComplex(fb_vec[i_e] * sum_xi, 0.0e0);
         H_out[ind] = hipCadd(H_out[ind],aux1);
      }

      aux1 = make_hipDoubleComplex(Efield, 0.0e0);
      aux1 = hipCmul(aux1, mu_tot[ind]);

      aux2 = make_hipDoubleComplex(sum_xi*v_bath_mat[ind], 0.0e0);

      aux3 = hipCadd(aux1,aux2);
      H_out[ind] = hipCadd(H_out[ind], aux3);
   }
   return;
}
//##############################################################################
//This function extract the diagonal terms of the matrix matA in vecA
__global__ void get_diag(hipDoubleComplex *matA, hipDoubleComplex *vecA,
                         int n_tot){

   int ind  = threadIdx.x + blockIdx.x * blockDim.x;
   int dim2 = n_tot  * n_tot;
   int i1   = ind / n_tot;
   if ((ind == i1 + i1*n_tot) && (ind < dim2)){
      vecA[i1] = matA[ind];
   }
   return;
}
//##############################################################################
__global__ void build_rhophon(hipDoubleComplex *rho_tot,
                              hipDoubleComplex *rho_phon, int n_el ,int n_phon,
                              int np_levels, int n_tot){

   int ind1 = threadIdx.x + blockIdx.x * blockDim.x;
   int dim1 = n_phon * np_levels;
   int dim2 = dim1 * dim1;

   if (ind1 < dim2){
      int jj = ind1/dim1;
      int ii = ind1 - jj * dim1;
      rho_phon[ind1] = make_hipDoubleComplex(0.0e0, 0.0e0);
      for (int kk=0; kk<n_el; kk++){
         int ind2 = (ii + kk * dim1) + (jj + kk * dim1) * n_tot;
         rho_phon[ind1] = hipCadd(rho_tot[ind2], rho_phon[ind1]);
      }
   }
   return;
}
//##############################################################################
__global__ void move_x(double *xi_vec, double *vi_vec, double *xf_vec,
                           double dt, int n_bath){
   int ind        = threadIdx.x + blockIdx.x * blockDim.x;
   if (ind < n_bath){
      xf_vec[ind]  = xi_vec[ind] + vi_vec[ind] * dt;
   }
   return;
}
//##############################################################################
__global__ void get_partial_sum(double *xi_vec, double *sum_vec, int n_bath){
   __shared__ double cache[Nthreads];
   int ind        = threadIdx.x + blockIdx.x * blockDim.x;
   int cacheIndex = threadIdx.x;

   cache[cacheIndex] = 0.0e0;
   if (ind < n_bath){
      cache[cacheIndex] = xi_vec[ind];
      __syncthreads();
      int ii = blockDim.x/2;
      while (ii != 0) {
         if (cacheIndex < ii){
            cache[cacheIndex] += cache[cacheIndex + ii];
         }
         __syncthreads();
         ii /= 2;
      }
      if (cacheIndex == 0){
         sum_vec[blockIdx.x] = cache[0];
      }
   }
   return;
}
//##############################################################################
__global__ void get_partial_Ek(double *vi_vec, double *sum_vec, int n_bath){
   __shared__ double cache[Nthreads];
   int ind        = threadIdx.x + blockIdx.x * blockDim.x;
   int cacheIndex = threadIdx.x;

   cache[cacheIndex] = 0.0e0;
   if (ind < n_bath){
      cache[cacheIndex] = vi_vec[ind]*vi_vec[ind];
      __syncthreads();
      int ii = blockDim.x/2;
      while (ii != 0) {
         if (cacheIndex < ii){
            cache[cacheIndex] += cache[cacheIndex + ii];
         }
         __syncthreads();
         ii /= 2;
      }
      if (cacheIndex == 0){
         sum_vec[blockIdx.x] = cache[0];
      }
   }
   return;
}
//##############################################################################
__global__ void move_v(double *xi_vec, double *vi_vec, double *ki_vec,
                       double *vf_vec, double qforce, double dt,
                       int n_bath){

   int ind  = threadIdx.x + blockIdx.x * blockDim.x;
   if (ind < n_bath){
      double ai = - ki_vec[ind] * xi_vec[ind] + qforce;
      vf_vec[ind]  = vi_vec[ind] + ai * dt;
   }
}
//##############################################################################
__global__ void update_mat(hipDoubleComplex *matA, hipDoubleComplex *matB,
                           int dim){
   int ind  = threadIdx.x + blockIdx.x * blockDim.x;
   int dim2 = dim * dim;
   if (ind < dim2){
      matA[ind] = matB[ind];
   }
   return;
}
//##############################################################################
__global__ void update_vec(double *vecA, double *vecB, int dim){
   int ind  = threadIdx.x + blockIdx.x * blockDim.x;
   if (ind < dim){
      vecA[ind] = vecB[ind];
   }
   return;
}
//##############################################################################
__global__ void get_long_ke_vectors(hipDoubleComplex *rho, double *ke_N_phon,
                                    double *eta_long, double *lambda_long,
                                    double *ke_del1, double *ke_del2,
                                    int *ke_ind_j, int *ke_ind_k,
                                    int n_tot, int n_ke_inter){

   int ind     = threadIdx.x + blockIdx.x * blockDim.x;

   if (ind < n_ke_inter){
      int jj      = ke_ind_j[ind];
      int kk      = ke_ind_k[ind];
      double del1 = ke_del1[ind];
      double del2 = ke_del2[ind];
      double f2   = hipCreal(rho[jj + jj * n_tot]);
      double Nj   = ke_N_phon[kk];

      eta_long[ind]    = -((Nj + f2) * del1 + (Nj - f2 + 1.0e0) * del2);
      lambda_long[ind] = f2*((Nj + 1.0e0) * del1 + Nj * del2);
   }

   return;
}
//##############################################################################
__global__ void apply_ke_term(hipDoubleComplex *rho, hipDoubleComplex *Drho,
                              double *eta_short, double *lambda_short,
                              int n_tot){

   int ind  = threadIdx.x + blockIdx.x * blockDim.x;
   int dim2 = n_tot  * n_tot;
   int i1   = ind / n_tot;
   int i2   = ind - i1*n_tot;
   if ((ind == i2 + i1*n_tot) && (ind < dim2)){
   // if ((ind == i1 + i1*n_tot) && (ind < dim2)){
      hipDoubleComplex aux1 = make_hipDoubleComplex(eta_short[i1], 0.0e0);
      hipDoubleComplex aux2 = make_hipDoubleComplex(eta_short[i2], 0.0e0);
      hipDoubleComplex aux3;
      hipDoubleComplex aux4 = make_hipDoubleComplex(0.5e0, 0.0e0);
      hipDoubleComplex aux5 = make_hipDoubleComplex(0.0e0,0.0e0);
      aux3 = hipCadd(aux1,aux2);
      aux3 = hipCmul(aux4,aux3);
      aux3 = hipCmul(aux3,rho[i2+i1*n_tot]);
      if (ind == i1 + i1*n_tot){
         aux5 = make_hipDoubleComplex(lambda_short[i1], 0.0e0);
      }
      aux5 = hipCadd(aux3,aux5);
      Drho[i2+i1*n_tot] = hipCadd(Drho[i2+i1*n_tot], aux5);
   }
   return;
}

//##############################################################################
void init_cuda(complex<double> *H_tot, complex<double> *mu_tot,
               double *v_bath_mat, double *fb_vec, double *xi_vec,
               double *vi_vec, double *ki_vec,
               complex<double> *rho_tot,
               complex<double> *rho_phon, complex<double> *dVdX_mat,
               int *ke_index_i, int *ke_index_j, int *ke_index_k,
               double *ke_delta1_vec, double *ke_delta2_vec,
               double *ke_N_phon_vec,
               UNINT n_el, UNINT n_phon, UNINT np_levels, UNINT n_tot,
               UNINT n_bath, UNINT n_ke_bath, UNINT n_ke_inter){

   double gaux = (double) (n_tot*n_tot);
   double taux = (double) Nthreads;

   Ncores1 = (UNINT) ceil(gaux/taux);
   gaux    = (double) (n_bath);
   Ncores2 = (UNINT) ceil(gaux/taux);
   gaux    = (double) (n_ke_inter);
   Ncores3 = (UNINT) ceil(gaux/taux);

   int dimaux  = n_phon * n_phon * np_levels * np_levels;

   hipMalloc((void**) &dev_Htot1  , n_tot*n_tot*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_Htot2  , n_tot*n_tot*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_Htot3  , n_tot*n_tot*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_mutot  , n_tot*n_tot*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_rhotot , n_tot*n_tot*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_rhonew , n_tot*n_tot*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_rhoaux , n_tot*n_tot*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_Drho   , n_tot*n_tot*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_rhophon, dimaux*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_dvdx   , dimaux*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_vbath     , n_tot*n_tot*sizeof(double));
   hipMalloc((void**) &dev_fb        , n_el*sizeof(double));
   hipMalloc((void**) &dev_xi        , n_bath*sizeof(double));
   hipMalloc((void**) &dev_vi        , n_bath*sizeof(double));
   hipMalloc((void**) &dev_ki        , n_bath*sizeof(double));
   hipMalloc((void**) &dev_xf        , n_bath*sizeof(double));
   hipMalloc((void**) &dev_vf        , n_bath*sizeof(double));
   hipMalloc((void**) &dev_xh        , n_bath*sizeof(double));
   hipMalloc((void**) &dev_etaL_ke   , n_ke_inter*sizeof(double));
   hipMalloc((void**) &dev_lambdaL_ke, n_ke_inter*sizeof(double));
   hipMalloc((void**) &dev_etaS_ke   , n_tot*sizeof(double));
   hipMalloc((void**) &dev_lambdaS_ke, n_tot*sizeof(double));
   hipMalloc((void**) &dev_ke_del1   , n_ke_inter*sizeof(double));
   hipMalloc((void**) &dev_ke_del2   , n_ke_inter*sizeof(double));
   hipMalloc((void**) &dev_Nphon_ke  , n_ke_bath*sizeof(double));
   hipMalloc((void**) &dev_keind_j   , n_ke_inter*sizeof(int));
   hipMalloc((void**) &dev_keind_k   , n_ke_inter*sizeof(int));


   hipMemcpy(dev_Htot1, H_tot, n_tot*n_tot*sizeof(hipDoubleComplex),
              hipMemcpyHostToDevice);
   hipMemcpy(dev_mutot, mu_tot, n_tot*n_tot*sizeof(hipDoubleComplex),
              hipMemcpyHostToDevice);
   hipMemcpy(dev_rhotot, rho_tot, n_tot*n_tot*sizeof(hipDoubleComplex),
              hipMemcpyHostToDevice);
   hipMemcpy(dev_rhophon, rho_phon, dimaux*sizeof(hipDoubleComplex),
              hipMemcpyHostToDevice);
   hipMemcpy(dev_dvdx, dVdX_mat, dimaux*sizeof(hipDoubleComplex),
              hipMemcpyHostToDevice);
   hipMemcpy(dev_vbath, v_bath_mat, n_tot*n_tot*sizeof(double),
              hipMemcpyHostToDevice);
   hipMemcpy(dev_fb, fb_vec, n_el*sizeof(double), hipMemcpyHostToDevice);
   hipMemcpy(dev_xi, xi_vec, n_bath*sizeof(double), hipMemcpyHostToDevice);
   hipMemcpy(dev_vi, vi_vec, n_bath*sizeof(double), hipMemcpyHostToDevice);
   hipMemcpy(dev_ki, ki_vec, n_bath*sizeof(double), hipMemcpyHostToDevice);
   hipMemcpy(dev_ke_del1, ke_delta1_vec, n_ke_inter*sizeof(double),
              hipMemcpyHostToDevice);
   hipMemcpy(dev_ke_del2, ke_delta2_vec, n_ke_inter*sizeof(double),
              hipMemcpyHostToDevice);
   hipMemcpy(dev_Nphon_ke, ke_N_phon_vec, n_ke_bath*sizeof(double),
              hipMemcpyHostToDevice);
   hipMemcpy(dev_keind_j, ke_index_j, n_ke_inter*sizeof(int),
              hipMemcpyHostToDevice);
   hipMemcpy(dev_keind_k, ke_index_k, n_ke_inter*sizeof(int),
              hipMemcpyHostToDevice);

   return;
}
//##############################################################################
void free_cuda_memory(){

   hipFree(dev_Htot1);
   hipFree(dev_Htot2);
   hipFree(dev_Htot3);
   hipFree(dev_mutot);
   hipFree(dev_rhotot);
   hipFree(dev_rhonew);
   hipFree(dev_rhoaux);
   hipFree(dev_Drho);
   hipFree(dev_rhophon);
   hipFree(dev_dvdx);
   hipFree(dev_vbath);
   hipFree(dev_fb);
   hipFree(dev_ki);
   hipFree(dev_xi);
   hipFree(dev_xf);
   hipFree(dev_xh);
   hipFree(dev_vi);
   hipFree(dev_vf);
   hipFree(dev_etaL_ke);
   hipFree(dev_lambdaL_ke);
   hipFree(dev_etaS_ke);
   hipFree(dev_lambdaS_ke);
   hipFree(dev_ke_del1);
   hipFree(dev_ke_del2);
   hipFree(dev_Nphon_ke);
   hipFree(dev_keind_j);
   hipFree(dev_keind_k);

   return;
}
//##############################################################################
void matmul_cublas(hipDoubleComplex *dev_A, hipDoubleComplex *dev_B,
                   hipDoubleComplex *dev_C, int dim){

   const hipDoubleComplex alf = make_hipDoubleComplex(1.0,0.0);
   const hipDoubleComplex bet = make_hipDoubleComplex(0.0, 0.0);
   const hipDoubleComplex *alpha = &alf;
   const hipDoubleComplex *beta = &bet;
// Create a handle for CUBLAS
   hipblasHandle_t handle;
   hipblasCreate(&handle);
// Do the actual multiplication
    hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, alpha, dev_A,
                dim, dev_B, dim, beta, dev_C, dim);
// Destroy the handle
    hipblasDestroy(handle);
    return;

}
//##############################################################################
void commute_cuda(hipDoubleComplex *dev_A, hipDoubleComplex *dev_B,
                  hipDoubleComplex *dev_C, int dim, const hipDoubleComplex alf){

   const hipDoubleComplex bet1 = make_hipDoubleComplex(0.0, 0.0);
   const hipDoubleComplex bet2 = make_hipDoubleComplex(-1.0, 0.0);
   const hipDoubleComplex *alpha = &alf;
   const hipDoubleComplex *beta1 = &bet1;
   const hipDoubleComplex *beta2 = &bet2;
// Create a handle for CUBLAS
   hipblasHandle_t handle;
   hipblasCreate(&handle);
// Computing B.A
   hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, alpha, dev_B,
               dim, dev_A, dim, beta1, dev_C, dim);
// Computing A.B - B.A
   hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, alpha, dev_A,
               dim, dev_B, dim, beta2, dev_C, dim);
// Destroy the handle
   hipblasDestroy(handle);
   return;
}
//##############################################################################
void matadd_cublas(hipDoubleComplex *dev_A, hipDoubleComplex *dev_B,
                   hipDoubleComplex *dev_C, int dim, const hipDoubleComplex alf,
                   const hipDoubleComplex bet){

   const hipDoubleComplex *alpha = &alf;
   const hipDoubleComplex *beta = &bet;
// Create a handle for CUBLAS
   hipblasHandle_t handle;
   hipblasCreate(&handle);
// Do the actual multiplication
    hipblasZgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, alpha, dev_A,
                dim, beta ,dev_B, dim, dev_C, dim);
// Destroy the handle
    hipblasDestroy(handle);
    return;
}
//##############################################################################
double get_trace_cuda(hipDoubleComplex *dev_A, UNINT dim){

   // hipDoubleComplex aux1= make_hipDoubleComplex(0.0e0, 0.0e0);
   complex<double> aux1;
   double          aux2;
   complex<double> aux_vec[dim];
   hipDoubleComplex *dev_vec;

   hipMalloc((void**) &dev_vec, dim * sizeof(hipDoubleComplex));

   get_diag<<<Ncores1, Nthreads>>>(dev_A, dev_vec, dim);

   hipMemcpy(aux_vec, dev_vec, dim*sizeof(hipDoubleComplex),
              hipMemcpyDeviceToHost);

   for(int ii=0;ii<dim;ii++){
      aux1 += aux_vec[ii];
   }

   aux2 = aux1.real();

   hipFree(dev_vec);
   return aux2;
}
//##############################################################################
void include_Hceed_cuda(hipDoubleComplex *dev_Hout, hipDoubleComplex *dev_Hin,
                        hipDoubleComplex *dev_mu, hipDoubleComplex *dev_rhoin,
                        double a_ceed, int n_tot){

   int dim2 = n_tot*n_tot;
   double dmu2;
   hipDoubleComplex *dev_aux1, *dev_aux2, *dev_Hceed;
   const hipDoubleComplex alf  = make_hipDoubleComplex(1.0,0.0);

   hipMalloc((void**) &dev_aux1, dim2 * sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_aux2, dim2 * sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_Hceed, dim2 * sizeof(hipDoubleComplex));

   commute_cuda(dev_mu, dev_Hin, dev_Hceed, n_tot, alf);
   commute_cuda(dev_Hceed, dev_Hin, dev_aux1, n_tot, alf);
   matmul_cublas(dev_rhoin, dev_aux1, dev_aux2, n_tot);

   dmu2 = get_trace_cuda(dev_aux2, n_tot);

   const hipDoubleComplex bet = make_hipDoubleComplex(0.0, a_ceed*dmu2);

   matadd_cublas(dev_Hin, dev_Hceed, dev_Hout, n_tot, alf, bet);

   hipFree(dev_aux1);
   hipFree(dev_aux2);
   hipFree(dev_Hceed);

   return;
}
//##############################################################################
double get_Qforces_cuda(hipDoubleComplex *dev_rhoin ,double *fb_vec,
                        UNINT n_el, UNINT n_phon, UNINT np_levels, UNINT n_tot){


   UNINT            dim1   = np_levels * n_phon;
   UNINT            dim2   = dim1 * dim1;
   double           qforce = 0.0e0;
   complex<double>  aux_vec[n_tot];
   hipDoubleComplex *dev_vec, *dev_mat;

   hipMalloc((void**) &dev_vec, n_tot * sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_mat, dim2 * sizeof(hipDoubleComplex));

   get_diag<<<Ncores1, Nthreads>>>(dev_rhoin, dev_vec, n_tot);

   hipMemcpy(aux_vec, dev_vec, n_tot*sizeof(hipDoubleComplex),
              hipMemcpyDeviceToHost);

   for (int kk=0; kk<n_el; kk++){
   for (int ii=0; ii<dim1; ii++){
      qforce += -aux_vec[ii+kk*dim1].real() * fb_vec[kk];
   }
   }

   build_rhophon<<<Ncores1, Nthreads>>>(dev_rhoin, dev_rhophon, n_el , n_phon,
                                        np_levels, n_tot);
   matmul_cublas(dev_rhophon, dev_dvdx, dev_mat, dim1);

   get_diag<<<Ncores1, Nthreads>>>(dev_mat, dev_vec, dim1);

   hipMemcpy(aux_vec, dev_vec, n_tot*sizeof(hipDoubleComplex),
              hipMemcpyDeviceToHost);

   for (int ii=0; ii<dim1; ii++){
      qforce += -aux_vec[ii].real();
   }

   hipFree(dev_vec);
   hipFree(dev_mat);
   return qforce;
}
//##############################################################################
void include_ke_terms(hipDoubleComplex *dev_rho, hipDoubleComplex *dev_drdt,
                      int *ke_index_i, double *eta_s_vec,
                      double *lambda_s_vec, double *eta_l_vec,
                      double *lambda_l_vec, UNINT n_tot, UNINT n_ke_inter){

      get_long_ke_vectors<<<Ncores3, Nthreads>>>(dev_rho, dev_Nphon_ke,
                            dev_etaL_ke, dev_lambdaL_ke, dev_ke_del1,
                            dev_ke_del2, dev_keind_j, dev_keind_k,
                            n_tot, n_ke_inter);

      hipMemcpy(eta_l_vec, dev_etaL_ke, n_ke_inter*sizeof(double),
                 hipMemcpyDeviceToHost);
      hipMemcpy(lambda_l_vec, dev_lambdaL_ke, n_ke_inter*sizeof(double),
                 hipMemcpyDeviceToHost);

      for(int ii=0; ii<n_tot; ii++){
         eta_s_vec[ii] = 0.0e0;
         lambda_s_vec[ii] = 0.0e0;
      }

      for(int ii=0; ii<n_ke_inter; ii++){
         int ind_i = ke_index_i[ii];

         eta_s_vec[ind_i]    += eta_l_vec[ii];
         lambda_s_vec[ind_i] += lambda_l_vec[ii];
      }

      // printf("%f",lambda_s_vec[11]);

      hipMemcpy(dev_etaS_ke, eta_s_vec, n_tot*sizeof(double),
                 hipMemcpyHostToDevice);
      hipMemcpy(dev_lambdaS_ke, lambda_s_vec, n_tot*sizeof(double),
                 hipMemcpyHostToDevice);

      apply_ke_term<<<Ncores1, Nthreads>>>(dev_rho, dev_drdt, dev_etaS_ke,
                                           dev_lambdaS_ke, n_tot);

      return;
}
//##############################################################################
void runge_kutta_propagator_cuda(double mass_bath, double a_ceed, double dt,
                                 double Efield, double Efieldaux,
                                 double *fb_vec, double *eta_s_vec,
                                 double *lambda_s_vec, double *eta_l_vec,
                                 double *lambda_l_vec, int *ke_index_i,
                                 int tt, UNINT n_el,
                                 UNINT n_phon, UNINT np_levels,
                                 UNINT n_tot, UNINT n_bath, UNINT n_ke_inter){

   const hipDoubleComplex alf1 = make_hipDoubleComplex(0.5*dt,0.0e0);
   const hipDoubleComplex alf2 = make_hipDoubleComplex(dt, 0.0e0);
   const hipDoubleComplex alf3 = make_hipDoubleComplex(1.0e0, 0.0e0);
   const hipDoubleComplex alf4 = make_hipDoubleComplex(0.0e0, -1.0e0);
   double *dev_partialvec;
   double  partialvec[Ncores2];
   double  sum_xi;
   double  dth    = 0.5e0 * dt;
   double qforce;
   //double time = dt * tt;

   hipMalloc((void**) &dev_partialvec, Ncores2*sizeof(double));

   //Calculating the sum of all the coordintes of the bath----------------------
   get_partial_sum<<<Ncores2, Nthreads>>>(dev_xi, dev_partialvec, n_bath);

   hipMemcpy(partialvec, dev_partialvec, Ncores2*sizeof(double),
              hipMemcpyDeviceToHost);

   sum_xi = 0.0e0;
   for (int ii=1; ii<Ncores2; ii++){
      sum_xi += partialvec[ii];
   }
   //---------------------------------------------------------------------------

   //Efield_t = Efield * exp(-pow(((time-10.0)/0.2),2.0));

   //Building the new Hamiltonian at time = t ----------------------------------
   update_H_tot<<<Ncores1, Nthreads>>>(dev_Htot2, dev_Htot1, dev_mutot,
                                       dev_vbath, dev_fb, sum_xi, Efield,
                                       n_el, n_phon, np_levels, n_tot);
   //Including CEED Hamiltonian:
   include_Hceed_cuda(dev_Htot3, dev_Htot2, dev_mutot, dev_rhotot, a_ceed,
                      n_tot);
   //---------------------------------------------------------------------------

   //Calculating rho(t+dt/2) using LvN------------------------------------------
   commute_cuda(dev_Htot3, dev_rhotot, dev_Drho, n_tot, alf4);
   include_ke_terms(dev_rhotot, dev_Drho, ke_index_i, eta_s_vec, lambda_s_vec,
                    eta_l_vec, lambda_l_vec, n_tot, n_ke_inter);
   matadd_cublas(dev_rhotot, dev_Drho, dev_rhoaux, n_tot, alf3, alf1);
   //---------------------------------------------------------------------------
   //Calculating x(t+dt/2) and v(t+dt/2) using the Quantum forces --------------
   qforce = get_Qforces_cuda(dev_rhotot , fb_vec, n_el, n_phon, np_levels,
                             n_tot);
   qforce = qforce/mass_bath;
   move_x<<<Ncores2, Nthreads>>>(dev_xi, dev_vi, dev_xh, dth, n_bath);


   move_v<<<Ncores2, Nthreads>>>(dev_xi, dev_vi, dev_ki, dev_vf, qforce, dth,
                                 n_bath);
   //---------------------------------------------------------------------------
   //Hencefort we repeat everything to obtain everything in t + dt -------------

   get_partial_sum<<<Ncores2, Nthreads>>>(dev_xh, dev_partialvec, n_bath);

   hipMemcpy(partialvec, dev_partialvec, Ncores2*sizeof(double),
   hipMemcpyDeviceToHost);

   sum_xi = 0.0e0;
   for (int ii=1; ii<Ncores2; ii++){
      sum_xi += partialvec[ii];
   }

   //Efield_t = Efield * exp(-pow(((time+dth-10.0)/0.2),2.0));

   update_H_tot<<<Ncores1, Nthreads>>>(dev_Htot2, dev_Htot1, dev_mutot,
                                       dev_vbath, dev_fb, sum_xi, Efieldaux,
                                       n_el, n_phon, np_levels, n_tot);

   include_Hceed_cuda(dev_Htot3, dev_Htot2, dev_mutot, dev_rhoaux, a_ceed,
                      n_tot);

   commute_cuda(dev_Htot3, dev_rhoaux, dev_Drho, n_tot, alf4);
   include_ke_terms(dev_rhoaux, dev_Drho, ke_index_i, eta_s_vec, lambda_s_vec,
                    eta_l_vec, lambda_l_vec, n_tot, n_ke_inter);
   matadd_cublas(dev_rhotot, dev_Drho, dev_rhonew, n_tot, alf3, alf2);

   qforce = get_Qforces_cuda(dev_rhoaux , fb_vec, n_el, n_phon, np_levels,
                             n_tot);
   qforce = qforce/mass_bath;
   move_x<<<Ncores2, Nthreads>>>(dev_xi, dev_vf, dev_xf, dt, n_bath);

   move_v<<<Ncores2, Nthreads>>>(dev_xh, dev_vi, dev_ki, dev_vf, qforce, dt,
                                 n_bath);
   //---------------------------------------------------------------------------
   //We update rho, x and v:
   update_mat<<<Ncores1, Nthreads>>>(dev_rhotot, dev_rhonew, n_tot);
   update_vec<<<Ncores2, Nthreads>>>(dev_xi, dev_xf, n_bath);
   update_vec<<<Ncores2, Nthreads>>>(dev_vi, dev_vf, n_bath);

   hipFree(dev_partialvec);

   return;
}
//##############################################################################
void calcrhophon(hipDoubleComplex *dev_rhoin, int n_el, int n_phon,
                 int np_levels, int n_tot){
   build_rhophon<<<Ncores1, Nthreads>>>(dev_rhoin, dev_rhophon, n_el , n_phon,
                                        np_levels, n_tot);
   return;
}
//##############################################################################
void getingmat(complex<double> *matA, hipDoubleComplex *dev_A, int n_tot){

   hipMemcpy(matA, dev_A, n_tot*n_tot*sizeof(hipDoubleComplex),
              hipMemcpyDeviceToHost);
   return;
}
//##############################################################################
void getting_printing_info(double *Ener, double *mu, complex<double> *tr_rho,
                           double *Ek_bath, complex<double> *rho_tot,
                           UNINT n_tot, UNINT n_bath){

   int dim2 = n_tot * n_tot;
   hipDoubleComplex *dev_aux1;
   hipDoubleComplex *dev_vec;
   double *dev_partialvec;
   double  partialvec[Ncores2];

   hipMalloc((void**) &dev_aux1, dim2 * sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_vec, n_tot * sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_partialvec, Ncores2*sizeof(double));

   matmul_cublas(dev_rhotot, dev_Htot1, dev_aux1, n_tot);
   *Ener = get_trace_cuda(dev_aux1, n_tot);

   matmul_cublas(dev_rhotot, dev_mutot, dev_aux1, n_tot);
   *mu = get_trace_cuda(dev_aux1, n_tot);

   get_diag<<<Ncores1, Nthreads>>>(dev_rhotot, dev_vec, n_tot);

   hipMemcpy(tr_rho, dev_vec, n_tot*sizeof(hipDoubleComplex),
   hipMemcpyDeviceToHost);

   get_partial_Ek<<<Ncores2, Nthreads>>>(dev_vi, dev_partialvec, n_bath);
   hipMemcpy(partialvec, dev_partialvec, Ncores2*sizeof(double),
              hipMemcpyDeviceToHost);

   *Ek_bath = 0.0e0;
   for (int ii=1; ii<Ncores2; ii++){
      *Ek_bath += 0.5e0 * partialvec[ii];
   }

   hipMemcpy(rho_tot, dev_rhotot, n_tot*n_tot*sizeof(hipDoubleComplex),
   hipMemcpyDeviceToHost);

   hipFree(dev_vec);
   hipFree(dev_aux1);
   hipFree(dev_partialvec);

   return;
}
//##############################################################################
