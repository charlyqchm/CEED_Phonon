#include "hip/hip_runtime.h"
#include "cuda_subs.h"

hipDoubleComplex  *dev_rhophon;
hipDoubleComplex  *dev_rhotot;
hipDoubleComplex  *dev_rhonew;
hipDoubleComplex  *dev_rhoaux;
hipDoubleComplex  *dev_Drho;
hipDoubleComplex  *dev_Htot1;
hipDoubleComplex  *dev_Htot2;
hipDoubleComplex  *dev_Htot3;
hipDoubleComplex  *dev_mutot;
hipDoubleComplex  *dev_Xmat;
hipDoubleComplex  *dev_Pmat;
UNINT             Ncores1;
UNINT             Ncores2;
UNINT             Ncores3;

//##############################################################################
// This function build the Hamiltonian without CEED:
// H_in  = H_e + H_phon + H_e-phon - This matrix is the same always.
// H_out = H_in + E * mu + V_bath
__global__ void update_H_tot(hipDoubleComplex *H_out, hipDoubleComplex *H_in,
                             hipDoubleComplex *mu_tot, double Efield, int n_el,
                             int n_phon, int np_levels, int n_tot){

   int ind  = threadIdx.x + blockIdx.x * blockDim.x;
   int dim2 = n_tot * n_tot;
   hipDoubleComplex aux1;

   if (ind < dim2){
      H_out[ind] = H_in[ind];

      aux1 = make_hipDoubleComplex(Efield, 0.0e0);
      aux1 = hipCmul(aux1, mu_tot[ind]);

      H_out[ind] = hipCadd(H_out[ind], aux1);
   }
   return;
}
//##############################################################################
//This function extract the diagonal terms of the matrix matA in vecA
__global__ void get_diag(hipDoubleComplex *matA, hipDoubleComplex *vecA,
                         int n_tot){

   int ind  = threadIdx.x + blockIdx.x * blockDim.x;
   int dim2 = n_tot  * n_tot;
   int i1   = ind / n_tot;
   if ((ind == i1 + i1*n_tot) && (ind < dim2)){
      vecA[i1] = matA[ind];
   }
   return;
}
//##############################################################################
__global__ void build_rhophon(hipDoubleComplex *rho_tot,
                              hipDoubleComplex *rho_phon, int n_el ,int n_phon,
                              int np_levels, int n_tot){

   int ind1 = threadIdx.x + blockIdx.x * blockDim.x;
   int dim1 = n_phon * np_levels;
   int dim2 = dim1 * dim1;

   if (ind1 < dim2){
      int jj = ind1/dim1;
      int ii = ind1 - jj * dim1;
      rho_phon[ind1] = make_hipDoubleComplex(0.0e0, 0.0e0);
      for (int kk=0; kk<n_el; kk++){
         int ind2 = (ii + kk * dim1) + (jj + kk * dim1) * n_tot;
         rho_phon[ind1] = hipCadd(rho_tot[ind2], rho_phon[ind1]);
      }
   }
   return;
}
//##############################################################################
__global__ void update_mat(hipDoubleComplex *matA, hipDoubleComplex *matB,
                           int dim){
   int ind  = threadIdx.x + blockIdx.x * blockDim.x;
   int dim2 = dim * dim;
   if (ind < dim2){
      matA[ind] = matB[ind];
   }
   return;
}
//##############################################################################
void init_cuda(complex<double> *H_tot, complex<double> *mu_tot,
               complex<double> *rho_tot,
               complex<double> *rho_phon,
               complex<double> *X_phon_mat,
               complex<double> *P_phon_mat,
               UNINT n_el, UNINT n_phon, UNINT np_levels, UNINT n_tot){

   double gaux = (double) (n_tot*n_tot);
   double taux = (double) Nthreads;

   Ncores1 = (UNINT) ceil(gaux/taux);

   int dimaux  = n_phon * n_phon * np_levels * np_levels;

   hipMalloc((void**) &dev_Htot1  , n_tot*n_tot*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_Htot2  , n_tot*n_tot*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_Htot3  , n_tot*n_tot*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_mutot  , n_tot*n_tot*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_rhotot , n_tot*n_tot*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_rhonew , n_tot*n_tot*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_rhoaux , n_tot*n_tot*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_Drho   , n_tot*n_tot*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_Xmat   , n_tot*n_tot*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_Pmat   , n_tot*n_tot*sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_rhophon, dimaux*sizeof(hipDoubleComplex));

   hipMemcpy(dev_Htot1, H_tot, n_tot*n_tot*sizeof(hipDoubleComplex),
              hipMemcpyHostToDevice);
   hipMemcpy(dev_mutot, mu_tot, n_tot*n_tot*sizeof(hipDoubleComplex),
              hipMemcpyHostToDevice);
   hipMemcpy(dev_rhotot, rho_tot, n_tot*n_tot*sizeof(hipDoubleComplex),
              hipMemcpyHostToDevice);
   hipMemcpy(dev_rhophon, rho_phon, dimaux*sizeof(hipDoubleComplex),
              hipMemcpyHostToDevice);
   hipMemcpy(dev_Xmat, X_phon_mat, n_tot*n_tot*sizeof(hipDoubleComplex),
              hipMemcpyHostToDevice);
   hipMemcpy(dev_Pmat, P_phon_mat, n_tot*n_tot*sizeof(hipDoubleComplex),
              hipMemcpyHostToDevice);

   return;
}
//##############################################################################
void free_cuda_memory(){

   hipFree(dev_Htot1);
   hipFree(dev_Htot2);
   hipFree(dev_Htot3);
   hipFree(dev_mutot);
   hipFree(dev_rhotot);
   hipFree(dev_rhonew);
   hipFree(dev_rhoaux);
   hipFree(dev_Drho);
   hipFree(dev_rhophon);
   hipFree(dev_Xmat);
   hipFree(dev_Pmat);

   return;
}
//##############################################################################
void matmul_cublas(hipDoubleComplex *dev_A, hipDoubleComplex *dev_B,
                   hipDoubleComplex *dev_C, int dim){

   const hipDoubleComplex alf = make_hipDoubleComplex(1.0,0.0);
   const hipDoubleComplex bet = make_hipDoubleComplex(0.0, 0.0);
   const hipDoubleComplex *alpha = &alf;
   const hipDoubleComplex *beta = &bet;
// Create a handle for CUBLAS
   hipblasHandle_t handle;
   hipblasCreate(&handle);
// Do the actual multiplication
    hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, alpha, dev_A,
                dim, dev_B, dim, beta, dev_C, dim);
// Destroy the handle
    hipblasDestroy(handle);
    return;

}
//##############################################################################
void commute_cuda(hipDoubleComplex *dev_A, hipDoubleComplex *dev_B,
                  hipDoubleComplex *dev_C, int dim, const hipDoubleComplex alf){

   const hipDoubleComplex bet1 = make_hipDoubleComplex(0.0, 0.0);
   const hipDoubleComplex bet2 = make_hipDoubleComplex(-1.0, 0.0);
   const hipDoubleComplex *alpha = &alf;
   const hipDoubleComplex *beta1 = &bet1;
   const hipDoubleComplex *beta2 = &bet2;
// Create a handle for CUBLAS
   hipblasHandle_t handle;
   hipblasCreate(&handle);
// Computing B.A
   hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, alpha, dev_B,
               dim, dev_A, dim, beta1, dev_C, dim);
// Computing A.B - B.A
   hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, alpha, dev_A,
               dim, dev_B, dim, beta2, dev_C, dim);
// Destroy the handle
   hipblasDestroy(handle);
   return;
}
//##############################################################################
void anticommute_cuda(hipDoubleComplex *dev_A, hipDoubleComplex *dev_B,
                      hipDoubleComplex *dev_C, int dim,
                      const hipDoubleComplex alf){

   const hipDoubleComplex bet1 = make_hipDoubleComplex(0.0, 0.0);
   const hipDoubleComplex bet2 = make_hipDoubleComplex(1.0, 0.0);
   const hipDoubleComplex *alpha = &alf;
   const hipDoubleComplex *beta1 = &bet1;
   const hipDoubleComplex *beta2 = &bet2;
// Create a handle for CUBLAS
   hipblasHandle_t handle;
   hipblasCreate(&handle);
// Computing B.A
   hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, alpha, dev_B,
               dim, dev_A, dim, beta1, dev_C, dim);
// Computing A.B - B.A
   hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, dim, alpha, dev_A,
               dim, dev_B, dim, beta2, dev_C, dim);
// Destroy the handle
   hipblasDestroy(handle);
   return;
}
//##############################################################################
void matadd_cublas(hipDoubleComplex *dev_A, hipDoubleComplex *dev_B,
                   hipDoubleComplex *dev_C, int dim, const hipDoubleComplex alf,
                   const hipDoubleComplex bet){

   const hipDoubleComplex *alpha = &alf;
   const hipDoubleComplex *beta = &bet;
// Create a handle for CUBLAS
   hipblasHandle_t handle;
   hipblasCreate(&handle);
// Do the actual addition
    hipblasZgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, dim, alpha, dev_A,
                dim, beta ,dev_B, dim, dev_C, dim);
// Destroy the handle
    hipblasDestroy(handle);
    return;
}
//##############################################################################
double get_trace_cuda(hipDoubleComplex *dev_A, UNINT dim){

   // hipDoubleComplex aux1= make_hipDoubleComplex(0.0e0, 0.0e0);
   complex<double> aux1;
   double          aux2;
   complex<double> aux_vec[dim];
   hipDoubleComplex *dev_vec;

   hipMalloc((void**) &dev_vec, dim * sizeof(hipDoubleComplex));

   get_diag<<<Ncores1, Nthreads>>>(dev_A, dev_vec, dim);

   hipMemcpy(aux_vec, dev_vec, dim*sizeof(hipDoubleComplex),
              hipMemcpyDeviceToHost);

   for(int ii=0;ii<dim;ii++){
      aux1 += aux_vec[ii];
   }

   aux2 = aux1.real();

   hipFree(dev_vec);
   return aux2;
}
//##############################################################################
void include_Hceed_cuda(hipDoubleComplex *dev_Hout, hipDoubleComplex *dev_Hin,
                        hipDoubleComplex *dev_mu, hipDoubleComplex *dev_rhoin,
                        double a_ceed, int n_tot){

   int dim2 = n_tot*n_tot;
   double dmu2;
   hipDoubleComplex *dev_aux1, *dev_aux2, *dev_Hceed;
   const hipDoubleComplex alf  = make_hipDoubleComplex(1.0,0.0);

   hipMalloc((void**) &dev_aux1, dim2 * sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_aux2, dim2 * sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_Hceed, dim2 * sizeof(hipDoubleComplex));

   commute_cuda(dev_mu, dev_Hin, dev_Hceed, n_tot, alf);
   commute_cuda(dev_Hceed, dev_Hin, dev_aux1, n_tot, alf);
   matmul_cublas(dev_rhoin, dev_aux1, dev_aux2, n_tot);

   dmu2 = get_trace_cuda(dev_aux2, n_tot);

   const hipDoubleComplex bet = make_hipDoubleComplex(0.0, a_ceed*dmu2);

   matadd_cublas(dev_Hin, dev_Hceed, dev_Hout, n_tot, alf, bet);

   hipFree(dev_aux1);
   hipFree(dev_aux2);
   hipFree(dev_Hceed);

   return;
}
//##############################################################################
void include_noise_dumping(hipDoubleComplex *dev_rho, hipDoubleComplex *dev_drdt,
                           double LM_term, double C_term, int n_tot){

   int dim2 = n_tot*n_tot;
   hipDoubleComplex *dev_aux1, *dev_auxC, *dev_auxL;
   const hipDoubleComplex alf1   = make_hipDoubleComplex(1.0,0.0);
   const hipDoubleComplex alf2   = make_hipDoubleComplex(0.5,0.0);
   const hipDoubleComplex alf_C  = make_hipDoubleComplex(-C_term, 0.00);
   const hipDoubleComplex alf_L  = make_hipDoubleComplex(0.00, -LM_term);

   hipMalloc((void**) &dev_aux1, dim2 * sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_auxC, dim2 * sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_auxL, dim2 * sizeof(hipDoubleComplex));

//Calculating -C/\hbar^2 [X,[X,\rho]]
   commute_cuda(dev_Xmat, dev_rho, dev_aux1, n_tot, alf1);
   commute_cuda(dev_Xmat, dev_aux1, dev_auxC, n_tot, alf_C);

//Calculating -i L/ (\hbar M) [X, 0.5 {P, \rho}]
   anticommute_cuda(dev_Pmat, dev_rho, dev_aux1, n_tot, alf2);
   commute_cuda(dev_Xmat, dev_aux1, dev_auxL, n_tot, alf_L);

//Adding C and L term to d\rho/dt
   matadd_cublas(dev_drdt, dev_auxC, dev_aux1, n_tot, alf1, alf1);
   matadd_cublas(dev_aux1, dev_auxL, dev_drdt, n_tot, alf1, alf1);

   hipFree(dev_aux1);
   hipFree(dev_auxL);
   hipFree(dev_auxC);

   return;
}

//##############################################################################
void runge_kutta_propagator_cuda(double mass_bath, double a_ceed, double dt,
                                 double Efield, double Efieldaux,
                                 double C_term, double LM_term,
                                 int tt, UNINT n_el,
                                 UNINT n_phon, UNINT np_levels,
                                 UNINT n_tot){

   const hipDoubleComplex alf1 = make_hipDoubleComplex(0.5*dt,0.0e0);
   const hipDoubleComplex alf2 = make_hipDoubleComplex(dt, 0.0e0);
   const hipDoubleComplex alf3 = make_hipDoubleComplex(1.0e0, 0.0e0);
   const hipDoubleComplex alf4 = make_hipDoubleComplex(0.0e0, -1.0e0);
   //double time = dt * tt;

   //Efield_t = Efield * exp(-pow(((time-10.0)/0.2),2.0));

   //Building the new Hamiltonian at time = t ----------------------------------
   update_H_tot<<<Ncores1, Nthreads>>>(dev_Htot2, dev_Htot1, dev_mutot,
                                       Efield, n_el, n_phon, np_levels, n_tot);
   //Including CEED Hamiltonian:
   include_Hceed_cuda(dev_Htot3, dev_Htot2, dev_mutot, dev_rhotot, a_ceed,
                      n_tot);
   //---------------------------------------------------------------------------

   //Calculating rho(t+dt/2) using LvN------------------------------------------
   commute_cuda(dev_Htot3, dev_rhotot, dev_Drho, n_tot, alf4); // -i[H,\rho]
   // include_noise_dumping(dev_rhotot, dev_Drho, LM_term, C_term, n_tot);

   matadd_cublas(dev_rhotot, dev_Drho, dev_rhoaux, n_tot, alf3, alf1);
   //---------------------------------------------------------------------------
   //Hencefort we repeat everything to obtain everything in t + dt -------------

   //Efield_t = Efield * exp(-pow(((time+dth-10.0)/0.2),2.0));

   update_H_tot<<<Ncores1, Nthreads>>>(dev_Htot2, dev_Htot1, dev_mutot,
                                       Efieldaux, n_el, n_phon, np_levels,
                                       n_tot);

   include_Hceed_cuda(dev_Htot3, dev_Htot2, dev_mutot, dev_rhoaux, a_ceed,
                      n_tot);

   commute_cuda(dev_Htot3, dev_rhoaux, dev_Drho, n_tot, alf4);
   // include_noise_dumping(dev_rhoaux, dev_Drho, LM_term, C_term, n_tot);

   matadd_cublas(dev_rhotot, dev_Drho, dev_rhonew, n_tot, alf3, alf2);

   //---------------------------------------------------------------------------
   //We update rho, x and v:
   update_mat<<<Ncores1, Nthreads>>>(dev_rhotot, dev_rhonew, n_tot);

   return;
}
//##############################################################################
void calcrhophon(hipDoubleComplex *dev_rhoin, int n_el, int n_phon,
                 int np_levels, int n_tot){
   build_rhophon<<<Ncores1, Nthreads>>>(dev_rhoin, dev_rhophon, n_el , n_phon,
                                        np_levels, n_tot);
   return;
}
//##############################################################################
void getingmat(complex<double> *matA, hipDoubleComplex *dev_A, int n_tot){

   hipMemcpy(matA, dev_A, n_tot*n_tot*sizeof(hipDoubleComplex),
              hipMemcpyDeviceToHost);
   return;
}
//##############################################################################
void getting_printing_info(double *Ener, double *mu, complex<double> *tr_rho,
                           complex<double> *rho_tot, UNINT n_tot){

   int dim2 = n_tot * n_tot;
   hipDoubleComplex *dev_aux1;
   hipDoubleComplex *dev_vec;

   hipMalloc((void**) &dev_aux1, dim2 * sizeof(hipDoubleComplex));
   hipMalloc((void**) &dev_vec, n_tot * sizeof(hipDoubleComplex));

   matmul_cublas(dev_rhotot, dev_Htot1, dev_aux1, n_tot);
   *Ener = get_trace_cuda(dev_aux1, n_tot);

   matmul_cublas(dev_rhotot, dev_mutot, dev_aux1, n_tot);
   *mu = get_trace_cuda(dev_aux1, n_tot);

   get_diag<<<Ncores1, Nthreads>>>(dev_rhotot, dev_vec, n_tot);

   hipMemcpy(tr_rho, dev_vec, n_tot*sizeof(hipDoubleComplex),
   hipMemcpyDeviceToHost);

   // hipMemcpy(rho_tot, dev_rhotot, n_tot*n_tot*sizeof(hipDoubleComplex),
   // hipMemcpyDeviceToHost);

   hipFree(dev_vec);
   hipFree(dev_aux1);

   return;
}
//##############################################################################
